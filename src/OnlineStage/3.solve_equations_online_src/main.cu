#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//#include <helper_cuda.h>

#define DEBUG 0
#define UNKNOWN_NUM 188
#define POLY_NUM 191
#define PARAM_NUM 36

#define ENUM_NUM 16 //16
#define CHECK_NUM 9
#define SOL_MAX_NUM 200

//#define SET_VAL (((value_t)14764175488)<< ENUM_NUM)

#define BLOCK_NUM 32 //2^5
#define THREAD_NUM  256 // 2^8
#define THREADS_SHIFT 13 // (5+8)
typedef long value_t; // to save values of variables.

//typedef unsigned long constpart_t; // the part with no parameters.
typedef unsigned long linearpart_t; // to save 32 unknowns and 1 contants.
typedef unsigned long squarepart_t;

typedef unsigned long oripoly_t;



static inline void binary_print(value_t val, int len) {

    for (int i = 0; i < len; i++) {
        if (val & ((value_t)1 << i)) {
            printf("1");
        } else {
            printf("0");
        }

        if ((i + 1) % 5 == 0) {
            printf(" ");
        }
    }

}


typedef unsigned char UINT8;
typedef unsigned long long UINT64;
typedef UINT64 tKeccakLane;

//#define KeccakReference
#define maxNrRounds 24
#define nrLanes 25
#define index(x, y) (((x)%5)+5*((y)%5))
#define KeccakP1600_stateSizeInBytes    200


static tKeccakLane KeccakRoundConstants[maxNrRounds];
static unsigned int KeccakRhoOffsets[nrLanes];

__constant__ tKeccakLane const_KeccakRoundConstants[maxNrRounds] =
{
    0x0000000000000001,
    0x0000000000008082,
    0x800000000000808a,
    0x8000000080008000,
    0x000000000000808b,
    0x0000000080000001,
    0x8000000080008081,
    0x8000000000008009,
    0x000000000000008a,
    0x0000000000000088,
    0x0000000080008009,
    0x000000008000000a,
    0x000000008000808b,
    0x800000000000008b,
    0x8000000000008089,
    0x8000000000008003,
    0x8000000000008002,
    0x8000000000000080,
    0x000000000000800a,
    0x800000008000000a,
    0x8000000080008081,
    0x8000000000008080,
    0x0000000080000001,
    0x8000000080008008,
};

__constant__ unsigned int const_KeccakRhoOffsets[nrLanes] =
{
     0,  1, 62, 28, 27, 36, 44,  6, 55, 20,  3, 10, 43, 25, 39, 41, 45, 15, 21,  8, 18,  2, 61, 56, 14
};

void KeccakP1600_InitializeRoundConstants(void);
void KeccakP1600_InitializeRhoOffsets(void);
static int LFSR86540(UINT8 *LFSR);
int displayLevel = 10;

 void KeccakP1600_StaticInitialize(void)
{
    if (sizeof(tKeccakLane) != 8) {
        printf("tKeccakLane should be 64-bit wide\n");
        abort();
    }
    KeccakP1600_InitializeRoundConstants();
    KeccakP1600_InitializeRhoOffsets();
}

 void KeccakP1600_InitializeRoundConstants(void)
{
    UINT8 LFSRstate = 0x01;
    unsigned int i, j, bitPosition;

    for(i=0; i<maxNrRounds; i++) {
        KeccakRoundConstants[i] = 0;
        for(j=0; j<7; j++) {
            bitPosition = (1<<j)-1; /* 2^j-1 */
            if (LFSR86540(&LFSRstate))
                KeccakRoundConstants[i] ^= (tKeccakLane)1<<bitPosition;
        }
    }
}

 void KeccakP1600_InitializeRhoOffsets(void)
{
    unsigned int x, y, t, newX, newY;

    KeccakRhoOffsets[index(0, 0)] = 0;
    x = 1;
    y = 0;
    for(t=0; t<24; t++) {
        KeccakRhoOffsets[index(x, y)] = ((t+1)*(t+2)/2) % 64;
        newX = (0*x+1*y) % 5;
        newY = (2*x+3*y) % 5;
        x = newX;
        y = newY;
    }
}

static int LFSR86540(UINT8 *LFSR)
{
    int result = ((*LFSR) & 0x01) != 0;
    if (((*LFSR) & 0x80) != 0)
    /* Primitive polynomial over GF(2): x^8+x^6+x^5+x^4+1 */
        (*LFSR) = ((*LFSR) << 1) ^ 0x71;
    else
        (*LFSR) <<= 1;
    return result;
}

__host__ __device__ void KeccakP1600_Initialize(void *state)
{
    memset(state, 0, 1600/8);
}


/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_AddByte(void *state, unsigned char byte, unsigned int offset)
{
    assert(offset < 200);
    ((unsigned char *)state)[offset] ^= byte;
}

/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_AddBytes(void *state, const unsigned char *data, unsigned int offset, unsigned int length)
{
    unsigned int i;

    assert(offset < 200);
    assert(offset+length <= 200);
    for(i=0; i<length; i++)
        ((unsigned char *)state)[offset+i] ^= data[i];
}

/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_OverwriteBytes(void *state, const unsigned char *data, unsigned int offset, unsigned int length)
{
    assert(offset < 200);
    assert(offset+length <= 200);
    memcpy((unsigned char*)state+offset, data, length);
}

/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_OverwriteWithZeroes(void *state, unsigned int byteCount)
{
    assert(byteCount <= 200);
    memset(state, 0, byteCount);
}


#define ROL64(a, offset) ((offset != 0) ? ((((tKeccakLane)a) << offset) ^ (((tKeccakLane)a) >> (64-offset))) : a)

static __host__ __device__ void theta(tKeccakLane *A)
{
    unsigned int x, y;
    tKeccakLane C[5]={0,0,0,0,0}, D[5]={0,0,0,0,0};

    for(x=0; x<5; x++) {
        C[x] = 0;
        for(y=0; y<5; y++)
            C[x] ^= A[index(x, y)];
    }
    for(x=0; x<5; x++)
        D[x] = ROL64(C[(x+1)%5], 1) ^ C[(x+4)%5];
    for(x=0; x<5; x++)
        for(y=0; y<5; y++)
            A[index(x, y)] ^= D[x];
}

static void rho(tKeccakLane *A)
{
    unsigned int x, y;

    for(x=0; x<5; x++) for(y=0; y<5; y++)
        A[index(x, y)] = ROL64(A[index(x, y)], KeccakRhoOffsets[index(x, y)]);
}

static __device__ void rho_Device(tKeccakLane *A)
{
    unsigned int x, y;

    for(x=0; x<5; x++) for(y=0; y<5; y++)
        A[index(x, y)] = ROL64(A[index(x, y)], const_KeccakRhoOffsets[index(x, y)]);
}

static __host__ __device__ void pi(tKeccakLane *A)
{
    unsigned int x, y;
    tKeccakLane tempA[25];

    for(x=0; x<5; x++) for(y=0; y<5; y++)
        tempA[index(x, y)] = A[index(x, y)];
    for(x=0; x<5; x++) for(y=0; y<5; y++)
        A[index(0*x+1*y, 2*x+3*y)] = tempA[index(x, y)];
}

static __host__ __device__ void chi(tKeccakLane *A)
{
    unsigned int x, y;
    tKeccakLane C[5];

    for(y=0; y<5; y++) {
        for(x=0; x<5; x++)
            C[x] = A[index(x, y)] ^ ((~A[index(x+1, y)]) & A[index(x+2, y)]);
        for(x=0; x<5; x++)
            A[index(x, y)] = C[x];
    }
}

static void iota(tKeccakLane *A, unsigned int indexRound)
{
    A[index(0, 0)] ^= KeccakRoundConstants[indexRound];
}

static __device__ void iota_Device(tKeccakLane *A, unsigned int indexRound)
{
    A[index(0, 0)] ^= const_KeccakRoundConstants[indexRound];
}

void KeccakP1600Round(tKeccakLane *state, unsigned int indexRound)
{

    theta(state);
    rho(state);
    pi(state);
    chi(state);
    iota(state, indexRound);
}

__device__ void KeccakP1600Round_Device(tKeccakLane *state,unsigned int indexRound) {

	theta(state);
	rho_Device(state);
	pi(state);
	chi(state);
	iota_Device(state, indexRound);

}

void stateInit(tKeccakLane state[25]) {
	KeccakP1600_StaticInitialize();


	FILE *f = fopen("../data/state_files/messaged_state.txt", "r+"); //todo path
	char line[100];

	for (int k = 0; k < 5; k++) {
		fgets(line, 100, f);
		for (int i = 0; i < 5; i++) {
			char hex_s = 0;
			UINT64 tmp_val = 0;
			for (int j = 0; j < 17; j++) {

				if (line[i * 17 + j] <= '9' && line[i * 17 + j] >= '0') {
					hex_s = line[i * 17 + j] - '0';
				} else {
					hex_s = line[i * 17 + j] - 'a' + 10;
				}

				if (j < 16) {
					//printf("%c", line[i*17 + j] );
					tmp_val = (tmp_val << 4) ^ (hex_s);
				}

			}

			state[k * 5 + i] = tmp_val;
			//printf("%08x%08x ", (unsigned int) (state[k * 5 + i] >> 32),(unsigned int) (state[k * 5 + i] & 0xFFFFFFFFLU));
		}
	}

	fclose(f);

}

void getStates(tKeccakLane state[25], oripoly_t var_all[640][4], value_t val,
		value_t solutions[3]) {
	value_t val_sol[4];
	val_sol[3] = solutions[2];
	val_sol[2] = solutions[1];
	val_sol[1] = solutions[0];
	val_sol[0] = val ^ ((value_t) 1 << PARAM_NUM);

	for (int i = 0; i < 640; i++) {
		value_t w[4] = { 0, 0, 0, 0 };
		for (int j = 0; j < 4; j++) {
			w[j] = var_all[i][j] & val_sol[j];

		}

		w[0] = w[0] ^ w[1] ^ w[2] ^ w[3];
		w[0] = (w[0]) ^ (w[0] >> 32);
		w[0] = (w[0]) ^ (w[0] >> 16);
		w[0] = (w[0]) ^ (w[0] >> 8);
		w[0] = (w[0]) ^ (w[0] >> 4);
		w[0] = (w[0]) ^ (w[0] >> 2);
		w[0] = (w[0]) ^ (w[0] >> 1);
		if (w[0] & (value_t) 1) {
			int n = (i / 64 > 4) ? (i / 64 + 5) : i / 64;
			state[n] ^= ((UINT64) 1) << (i % 64);

		}
	}


}

int checkHashValue(tKeccakLane state[25], tKeccakLane hashvalue[4]) {


	tKeccakLane state_copy[25];
	for(int i = 0; i < 25; i++){
		state_copy[i] = state[i];
	}





	for (int i = 0; i < 3; i++) {
		KeccakP1600Round(state, i);
	}



	int result = 0;
	if(state[0] == hashvalue[0] && state[1] == hashvalue[1] && state[2] == hashvalue[2] && ((state[3] ^= hashvalue[3]) & (0x00000000FFFFFFFF)) == 0){
		printf("Find Preimage!!!\nState after XORed with block2:");
		for (int i = 0; i < 25; i++) {
			binary_print(state_copy[i], 64);
			printf(" ");
			printf("%llu ",state_copy[i]);
		}
		printf("\n");


		result = 1;
	}
	return result;
}




__constant__ tKeccakLane const_state[25];
__constant__ tKeccakLane const_hashvalue[4];


__device__ linearpart_t d_linear_mat[ENUM_NUM * POLY_NUM * 3];
__device__ squarepart_t d_square_mat[ENUM_NUM * POLY_NUM];
__device__ value_t d_var_all[2560];



static inline __host__ __device__ int largestpos(value_t val, int len) {

	for (int i = len - 1; i >= 0; i--) {
		if (val & ((value_t) 1 << i)) {
			return i;
		}
	}

	return -1;
}


static inline __host__ __device__ int largestpos_3(value_t val0, value_t val1,
		value_t val2, int len) {
	int p = 0;
	if (len > 128) {
		p = largestpos(val2, len - 128);
		if (p > -1) {
			return p + 128;
		} else {
			p = largestpos(val1, 64);
			if (p > -1) {
				return p + 64;
			} else {
				p = largestpos(val0, 64);

			}
		}
	} else if (len > 64 && len <= 128) {
		p = largestpos(val1, len - 64);
		if (p > -1) {
			return p + 64;
		} else {
			p = largestpos(val0, 64);

		}
	} else {
		p = largestpos(val0, 64);

	}

	return p;
}


static inline value_t gauss_host(linearpart_t working_mat[POLY_NUM][3],
		const int poly_num, const int unknown_num, value_t solutions[SOL_MAX_NUM][3]) {

	int pos_arr[POLY_NUM]; // bear revised
	int rank = 0;

	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 && working_mat[pi][2] == 0) {
			continue;
		}


		pos_arr[pi] = largestpos_3(working_mat[pi][0],working_mat[pi][1],working_mat[pi][2], unknown_num + 1);

		rank++;
		if (pos_arr[pi] == 0) {
			return 0;
		}





		for (int j = pi + 1; j < POLY_NUM; j++) {

			if(working_mat[j][pos_arr[pi]/64] & ((linearpart_t)1 << (pos_arr[pi] % 64))){
                working_mat[j][0] ^= (working_mat[pi][0]);
                working_mat[j][1] ^= (working_mat[pi][1]);
                working_mat[j][2] ^= (working_mat[pi][2]);
            }
		}



	}


	// back
	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
				&& working_mat[pi][2] == 0) {
			continue;
		}

		for (int j = 0; j < pi; j++) {
			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
				working_mat[j][2] ^= (working_mat[pi][2]);
			}
		}
	}

	if (rank == unknown_num) {

		// only one solution.
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 &&  working_mat[pi][2] == 0) {
				continue;
			}
			if (working_mat[pi][0] & (linearpart_t)1) {
			    solutions[0][(pos_arr[pi]-1) /64 ] ^= ((value_t)1 << (pos_arr[pi]-1) % 64);
			}
		}

		return 1;

	} else {

		// now troubles come
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		value_t sol_num = 1;
		bool appear[UNKNOWN_NUM + 1];
		for(int nn = 0; nn < UNKNOWN_NUM + 1; nn++){

			appear[nn] = 0;

		}



		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 && working_mat[pi][2] == 0) {
				continue;
			}

			appear[pos_arr[pi]] = true;
			if (working_mat[pi][0] & (linearpart_t)1) {
			    solutions[0][(pos_arr[pi]-1) /64 ] ^= ((value_t)1 << (pos_arr[pi]-1) % 64);
			}
		}

		// duplicate solutions.
		for (int i = 1; i < UNKNOWN_NUM+1; i++) {  // liting revised

			if (appear[i] == false) {


				for (int j = 0; j < sol_num; j++) {
				    // bear revised
					solutions[j + sol_num][0] = solutions[j][0];
					solutions[j + sol_num][1] = solutions[j][1];
					solutions[j + sol_num][2] = solutions[j][2];
					solutions[j + sol_num][(i-1)/64] ^= ((value_t)1 << ((i-1)%64));
				}

				// bear added
                for (int pi = 0; pi < POLY_NUM; pi++) {
				    if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 && working_mat[pi][2] == 0) {
			            continue;
		            }

				    for (int j = 0; j < sol_num * ((working_mat[pi][i/64] & (((linearpart_t) 1) << (i%64))) != 0); j++) {

				    	solutions[j + sol_num][(pos_arr[pi] - 1)/64] ^= ((value_t) 1 << ((pos_arr[pi] - 1)% 64));
				    }


				}


				sol_num *= 2;

			}
		}

		return sol_num;

	}

}




static inline __device__ value_t gauss(value_t solutions[SOL_MAX_NUM][3], linearpart_t working_mat[POLY_NUM][3],
		const int poly_num, const int unknown_num) {

	// bear revised
	int pos_arr[POLY_NUM]; // bear revised
	int rank = 0;

	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
				&& working_mat[pi][2] == 0) {
			continue;
		}

		pos_arr[pi] = largestpos_3(working_mat[pi][0], working_mat[pi][1],working_mat[pi][2],unknown_num + 1);
		rank++;


		if (pos_arr[pi] == 0) {
			return 0;
		}

		for (int j = pi + 1; j < POLY_NUM; j++) {

			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
				working_mat[j][2] ^= (working_mat[pi][2]);
			}
		}

	}

	// back
	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
				&& working_mat[pi][2] == 0) {
			continue;
		}

		for (int j = 0; j < pi; j++) {
			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
				working_mat[j][2] ^= (working_mat[pi][2]);
			}
		}
	}

	if (rank == unknown_num) {

		// only one solution.
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
					&& working_mat[pi][2] == 0) {
				continue;
			}
			if (working_mat[pi][0] & (linearpart_t) 1) {
				solutions[0][(pos_arr[pi] - 1) / 64] ^= ((value_t) 1
						<< (pos_arr[pi] - 1) % 64);
			}
		}

		return 1;

	} else {

		// now troubles come
		// now troubles come
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		value_t sol_num = 1;
		//liting revised
		bool appear[UNKNOWN_NUM + 1];
		for(int nn = 0; nn < UNKNOWN_NUM + 1; nn++){

			appear[nn] = 0;

		}

		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
					&& working_mat[pi][2] == 0) {
				continue;
			}

			appear[pos_arr[pi]] = true;
			if (working_mat[pi][0] & (linearpart_t) 1) {
				solutions[0][(pos_arr[pi] - 1) / 64] ^= ((value_t) 1
						<< (pos_arr[pi] - 1) % 64);
			}
		}

		// duplicate solutions.
		for (int i = 1; i < UNKNOWN_NUM + 1; i++) {  // liting revised

			if (appear[i] == false) {

				for (int j = 0; j < sol_num; j++) {
					// bear revised
					solutions[j + sol_num][0] = solutions[j][0];
					solutions[j + sol_num][1] = solutions[j][1];
					solutions[j + sol_num][2] = solutions[j][2];
					solutions[j + sol_num][(i - 1) / 64] ^= ((value_t) 1
							<< ((i - 1) % 64));
				}

				// bear added
				for (int pi = 0; pi < POLY_NUM; pi++) {
					if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
							&& working_mat[pi][2] == 0) {
						continue;
					}

					for (int j = 0;j< sol_num* ((working_mat[pi][i / 64]& (((linearpart_t) 1)<< (i % 64))) != 0);j++) {

						solutions[j + sol_num][(pos_arr[pi] - 1) / 64] ^=((value_t) 1 << ((pos_arr[pi] - 1) % 64));
					}

				}

				sol_num *= 2;

			}
		}

		return sol_num;

	}

}





__global__ void solveLinear(const linearpart_t *d_working_mat_copy,
		const squarepart_t *d_const_mat, value_t *d_val, value_t *d_sol_total,value_t* result) {

	int thidx = blockDim.x * blockIdx.x + threadIdx.x;
	value_t val = d_val[thidx];

	linearpart_t working_mat[POLY_NUM][3]; // initialized as the const part of linear matrix. also used as the results of linear part.
	linearpart_t working_mat_copy[POLY_NUM][3];

	squarepart_t const_mat[POLY_NUM];
	d_sol_total[thidx] = 0;



	//copy data from device
	for (int i = 0; i < POLY_NUM; i++) {
		working_mat_copy[i][0] = d_working_mat_copy[thidx * POLY_NUM * 3 + i*3];
		working_mat_copy[i][1] = d_working_mat_copy[thidx * POLY_NUM * 3 + i*3 + 1];
		working_mat_copy[i][2] = d_working_mat_copy[thidx * POLY_NUM * 3 + i*3 + 2];

		const_mat[i] = d_const_mat[thidx * POLY_NUM + i];
	}

	// main loop.
	for (value_t count = 1; count < (1 << ENUM_NUM); count++) {

		// generate the next gray code
		int pos = 64-__ffsll(__brevll(count ^ (count - 1)));

		val = val ^ ((value_t) 1 << pos);


		for (int pi = 0; pi < POLY_NUM; pi++) {
			working_mat_copy[pi][0] ^= d_linear_mat[pos * POLY_NUM * 3 + pi * 3];
			working_mat_copy[pi][1] ^= d_linear_mat[pos * POLY_NUM * 3 + pi * 3 + 1];
			working_mat_copy[pi][2] ^= d_linear_mat[pos * POLY_NUM * 3 + pi * 3 + 2];

			const_mat[pi] ^= d_square_mat[pos * POLY_NUM + pi];

			working_mat[pi][0] = working_mat_copy[pi][0];
			working_mat[pi][1] = working_mat_copy[pi][1];
			working_mat[pi][2] = working_mat_copy[pi][2];


			value_t w = const_mat[pi] & val;


			working_mat[pi][0] ^= (bool)((__popcll((unsigned long long int)w)) & (value_t) 1);


		}

		value_t solutions[SOL_MAX_NUM][3];
		value_t sol_num = 0;


		// gauss
		sol_num = gauss(solutions, working_mat, POLY_NUM, UNKNOWN_NUM);
		d_sol_total[thidx] += sol_num;
		// verify on 3 round keccak.
		tKeccakLane dState[25];


		for(int s = 0;s < sol_num;s++){
			dState[0] = 0;
			dState[1] = 0;
			dState[2] = 0;
			dState[3] = 0;
			dState[4] = 0;
			dState[5] = const_state[5];
			dState[6] = const_state[6];
			dState[7] = const_state[7];
			dState[8] = const_state[8];
			dState[9] = const_state[9];
			dState[10] = 0;
			dState[11] = 0;
			dState[12] = 0;
			dState[13] = 0;
			dState[14] = 0;
			dState[15] = const_state[15];
			dState[16] = const_state[16];
			dState[17] = const_state[17];
			dState[18] = const_state[18];
			dState[19] = const_state[19];

			dState[20] = const_state[20];
			dState[21] = const_state[21];
			dState[22] = const_state[22];
			dState[23] = const_state[23];
			dState[24] = const_state[24];


			value_t val_sol[4];
			val_sol[3] = solutions[s][2];
			val_sol[2] = solutions[s][1];
			val_sol[1] = solutions[s][0];
			val_sol[0] = val ^ ((value_t)1 << PARAM_NUM);

			for(int i = 0; i< 640; i ++){
				value_t w[4] ={0,0,0,0};
				for(int j = 0; j< 4; j++){
					w[j] = d_var_all[i * 4 + j] & val_sol[j];
				}

				w[0] = w[0] ^w[1]^w[2]^w[3];


				if ((bool)((__popcll((unsigned long long int)w[0])) & (value_t) 1)) {
					int n = (i/64 > 4 )?( i/64 + 5 ): i/64 ;
					dState[n] ^= ((UINT64)1) << (i % 64);
				}
			}


			tKeccakLane state_copy[25];
			for(int i = 0; i < 25; i++){
				state_copy[i] = dState[i];
			}



            KeccakP1600Round_Device(dState, 0);

            KeccakP1600Round_Device(dState, 1);
            KeccakP1600Round_Device(dState, 2);

            if(dState[0] == const_hashvalue[0] && dState[1] == const_hashvalue[1] && dState[2] == const_hashvalue[2] && ((dState[3] ^= const_hashvalue[3]) & (0x00000000FFFFFFFF)) == 0){
				printf("Find Preimage!!! val is %lu.\n",val);
				result[0] = val;
				result[1] = val_sol[1];
				result[2] = val_sol[2];
				result[3] = val_sol[3];
				printf("The messaged state :");
				for (int i = 0; i < 25; i++) {
					printf("%llx ", state_copy[i]);
					if(i % 5 == 0){
						printf("\n");
					}

				}
				printf("\n");
			}

		}
	}

}





int main(int argc, char** argv) {

	printf("read middle state\n");
	tKeccakLane state[25];
	stateInit(state);

	printf("read hash value\n");
   	FILE *hashvalue_file = fopen("../../hash_value.txt", "r+");
	tKeccakLane hashvalue[4] = { 0, 0, 0, 0 };
	char line[20];

	for (int i = 0; i < 4; i++) {
		fgets(line, 20, hashvalue_file);

		char hex_s = 0;
		UINT64 tmp_val = 0;
		for (int j = 0; j < 17; j++) {

			if (line[j] <= '9' && line[j] >= '0') {
				hex_s = line[j] - '0';
			}else if(line[j] <= 'z' && line[j] >= 'a'){
                hex_s = line[j] - 'a' + 10;
            }else if(line[j] <= 'Z' && line[j] >= 'A'){
                hex_s = line[j] - 'A' + 10;
            }

			if (j < 16) {
				//printf("%c", line[i*17 + j] );
				tmp_val = (tmp_val << 4) ^ (hex_s);
			}

		}

		hashvalue[i] = tmp_val;
	}
	fclose (hashvalue_file);

	hipError_t err0 = hipSuccess;
	err0 = hipMemcpyToSymbol(HIP_SYMBOL(const_state), state, 25 * sizeof(tKeccakLane), 0,
			hipMemcpyHostToDevice);
	if (err0 != hipSuccess) {
		printf("Failed to copy host to device(error code %s)!\n",
				hipGetErrorString(err0));
		exit(EXIT_FAILURE);
	}

	err0 = hipSuccess;
	err0 = hipMemcpyToSymbol(HIP_SYMBOL(const_hashvalue), hashvalue, 4 * sizeof(tKeccakLane), 0,
			hipMemcpyHostToDevice);
	if (err0 != hipSuccess) {
		printf("Failed to copy host to device(error code %s)!\n",
				hipGetErrorString(err0));
		exit(EXIT_FAILURE);
	}



	const int para_num = PARAM_NUM;
	const int enum_num = ENUM_NUM;
	//const int set_num = para_num - enum_num;

	value_t set_val = atol(argv[1])<<THREADS_SHIFT;
//	value_t set_val = 0;
	const int unknown_num = UNKNOWN_NUM;
	const int poly_num = POLY_NUM;

	linearpart_t linear_mat[para_num][poly_num][3];
	linearpart_t working_mat[poly_num][3]; // initialized as the const part of linear matrix. also used as the results of linear part.
	linearpart_t working_mat_copy[poly_num][3];
	linearpart_t working_mat_file[poly_num][3];

	squarepart_t square_mat[para_num][poly_num];
	squarepart_t const_mat[poly_num]; // used to compute the const part from square polys.
	oripoly_t var_all[640][4];

//	cudaSetDevice(0);
	hipSetDevice(atoi(argv[2]));

	//todo
	FILE *in1 = fopen("../data/mat_files/linear_mat.txt", "r+");
	FILE *in2 = fopen("../data/mat_files/square_mat.txt", "r+");
	FILE *in4 = fopen("../data/mat_files/working_mat.txt", "r+");
	FILE *in5 = fopen("../data/mat_files/totalLinear_mat.txt", "r+");

	char c1, c2, c4, c5;
	for (int i = 0; i < para_num; i++) {
		for (int j = 0; j < poly_num; j++) {
			linear_mat[i][j][0] = 0;
			linear_mat[i][j][1] = 0;
			linear_mat[i][j][2] = 0;
			square_mat[i][j] = 0;

			for (int k = 0; k < 192; k++) {
				fscanf(in1, "%c", &c1);
				while (c1 != '0' && c1 != '1') {
					fscanf(in1, "%c", &c1);
				}
				if (c1 == '1') {

					linear_mat[i][j][k / 64] ^= ((linearpart_t) 1 << (k % 64));
				}
			}

			for (int k = 0; k < 64; k++) {
				fscanf(in2, "%c", &c2);
				while (c2 != '0' && c2 != '1') {
					fscanf(in2, "%c", &c2);
				}
				if (c2 == '1') {
					square_mat[i][j] ^=
							((squarepart_t) 1 << (para_num - 1 - k));
				}
			}
		}

	}

	for (int i = 0; i < poly_num; i++) {
		working_mat[i][0] = 0;
		working_mat[i][1] = 0;
		working_mat[i][2] = 0;
		for (int j = 0; j < 192; j++) {
			fscanf(in4, "%c", &c4);
			while (c4 != '0' && c4 != '1') {
				fscanf(in4, "%c", &c4);
			}
			if (c4 == '1') {

				working_mat[i][j / 64] ^= ((linearpart_t) 1 << (j % 64));
			}
		}
		working_mat_file[i][0] = working_mat[i][0];
		working_mat_file[i][1] = working_mat[i][1];
		working_mat_file[i][2] = working_mat[i][2];
	}

	for (int i = 0; i < 640; i++) {
		var_all[i][0] = 0;
		var_all[i][1] = 0;
		var_all[i][2] = 0;
		var_all[i][3] = 0;
		for (int j = 0; j < 256; j++) {

			fscanf(in5, "%c", &c5);
			while (c5 != '0' && c5 != '1') {
				fscanf(in5, "%c", &c5);

			}
			if (c5 == '1') {
				var_all[i][j / 64] ^= ((value_t) 1 << (j % 64));
			}

		}

	}

	fclose(in1);
	fclose(in2);
	fclose(in4);
	fclose(in5);

	printf("finish reading file!\n");

	linearpart_t linear_mat_enum[ENUM_NUM * POLY_NUM * 3];
	squarepart_t square_mat_enum[ENUM_NUM * POLY_NUM];
	value_t var_all_enum[640 * 4];

	for (int i = 0; i < ENUM_NUM; i++) {
		for (int j = 0; j < POLY_NUM; j++) {
			for (int k = 0; k < 3; k++) {
				linear_mat_enum[i * POLY_NUM * 3 + j * 3 + k] =
						linear_mat[i][j][k];

			}
		}
	}

	for (int i = 0; i < ENUM_NUM; i++) {
		for (int j = 0; j < POLY_NUM; j++) {
			square_mat_enum[i * POLY_NUM + j] = square_mat[i][j];
		}
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_linear_mat), linear_mat_enum,
			3 * ENUM_NUM * POLY_NUM * sizeof(linearpart_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_square_mat), square_mat_enum,
			ENUM_NUM * POLY_NUM * sizeof(squarepart_t));

	for (int i = 0; i < 640; i++) {
		for (int j = 0; j < 4; j++) {
			var_all_enum[i * 4 + j] = var_all[i][j];
		}

	}
	hipMemcpyToSymbol(HIP_SYMBOL(d_var_all), var_all_enum, 640 * 4 * sizeof(value_t));

	printf("finish copying device memory!\n");

	hipError_t err = hipSuccess;
	int thidx = BLOCK_NUM * THREAD_NUM;

	value_t *d_val = NULL;
	err = hipMalloc((void **) &d_val, thidx * sizeof(value_t));
	if (err != hipSuccess) {
		printf("Failed to allocate device value (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	value_t *d_sol_total = NULL;
	err = hipMalloc((void **) &d_sol_total, thidx * 3 * sizeof(value_t));
	if (err != hipSuccess) {
		printf("Failed to allocate device value (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	linearpart_t *d_working_mat_copy = NULL;
	err = hipMalloc((void **) &d_working_mat_copy,
			thidx * poly_num * 3 * sizeof(linearpart_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate device working_mat_copy (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	squarepart_t *d_const_mat = NULL;
	err = hipMalloc((void **) &d_const_mat,
			thidx * poly_num * sizeof(squarepart_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate devices const_mat (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy oripolys from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	value_t h_result[4] = {0, 0, 0, 0};
	value_t *d_result = NULL;
	err = hipMalloc((void **) &d_result, 4 *sizeof(value_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate devices result (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_result, h_result,
	4 * sizeof(value_t),hipMemcpyHostToDevice);

		if (err != hipSuccess) {
			fprintf(stderr,
					"Failed to copy result from host to device (error code %s)!\n",
					hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}


	printf("finish allocate device memory!\n");

//	value_t val_arr[thidx];
//	linearpart_t working_mat_copy_arr[thidx * POLY_NUM * 3];
//	squarepart_t const_mat_arr[thidx * POLY_NUM];
	int s_total_p0 = 0;
	value_t *val_arr = (value_t*)calloc(thidx, sizeof(value_t));
	linearpart_t *working_mat_copy_arr = (linearpart_t*)calloc(thidx * POLY_NUM * 3, sizeof(linearpart_t));
	squarepart_t *const_mat_arr = (squarepart_t*)calloc(thidx * POLY_NUM, sizeof(squarepart_t));
	for (int thi = 0; thi < thidx; thi++) {

		value_t sol_num = 0;
		value_t solutions[SOL_MAX_NUM][3];

		//int sol_total = 0;
		value_t val = (set_val + (value_t) thi) << ENUM_NUM;
		val_arr[thi] = val;
		for (int pi = 0; pi < POLY_NUM; pi++) {
			working_mat[pi][0] = working_mat_file[pi][0];
			working_mat[pi][1] = working_mat_file[pi][1];
			working_mat[pi][2] = working_mat_file[pi][2];

			const_mat[pi] = 0;
		}

		for (int pos = enum_num; pos < para_num; pos++) {

			if (val & ((value_t) 1 << pos)) {

				for (int pi = 0; pi < poly_num; pi++) {
					working_mat[pi][0] ^= linear_mat[pos][pi][0];
					working_mat[pi][1] ^= linear_mat[pos][pi][1];
					working_mat[pi][2] ^= linear_mat[pos][pi][2];
				}

				for (int pi = 0; pi < poly_num; pi++) {
					const_mat[pi] ^= square_mat[pos][pi];

				}

			}

		}


		for (int i = 0; i < poly_num; i++) {
			working_mat_copy[i][0] = working_mat[i][0];
			working_mat_copy[i][1] = working_mat[i][1];
			working_mat_copy[i][2] = working_mat[i][2];

			working_mat_copy_arr[thi * POLY_NUM * 3 + 3 * i] = working_mat_copy[i][0];
			working_mat_copy_arr[thi * POLY_NUM * 3 + 3 * i + 1] = working_mat_copy[i][1];
			working_mat_copy_arr[thi * POLY_NUM * 3 + 3 * i + 2] = working_mat_copy[i][2];

			const_mat_arr[thi * POLY_NUM + i] = const_mat[i];

		}

		for (int pi = 0; pi < poly_num; pi++) {

			value_t w = const_mat[pi] & val;

			w = (w) ^ (w >> 32);
			w = (w) ^ (w >> 16);
			w = (w) ^ (w >> 8);
			w = (w) ^ (w >> 4);
			w = (w) ^ (w >> 2);
			w = (w) ^ (w >> 1);

			if (w & (value_t) 1) {

				working_mat[pi][0] ^= (linearpart_t) 1;
			}

		}

		sol_num = gauss_host(working_mat, POLY_NUM, UNKNOWN_NUM, solutions);

		s_total_p0 += sol_num;
		 for (int s = 0; s < sol_num; s++) {

			tKeccakLane state[25];
			stateInit(state);
			getStates(state, var_all, val, solutions[s]);

			tKeccakLane state_cp[25];
			for(int sn = 0; sn < 25; sn++){
				state_cp[sn] = state[sn];

			}
			if(checkHashValue(state, hashvalue)){
				//todo
				FILE *out = fopen("final_messaged_state_online.txt","a+");
				printf("we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",val,solutions[s][0],solutions[s][1],solutions[s][2]);
				//fprintf(out,"we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",val,solutions[s][0],solutions[s][1],solutions[s][2]);

				fprintf(out, "The messaged state:");
								for (int sn = 0; sn < 25; sn++) {
									if (sn % 5 == 0) {
										fprintf(out, "\n");

									}
									fprintf(out, "%016lX ", state_cp[sn]);

								}

				fprintf(out, "\n\nThe output state:");
				for(int sn = 0; sn < 25 ;sn++){

					if(sn % 5 == 0){
						fprintf(out, "\n");

					}
					fprintf(out, "%016lX ", state[sn]);
				}




				fclose(out);
				printf("finish.\n");
				exit(0);
			}



		    }


	}

	printf("finish cpu computing!\n");

	//begin device part
	err = hipMemcpy(d_val, val_arr, thidx * sizeof(value_t),
			hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("Failed to copy value from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_working_mat_copy, working_mat_copy_arr,
			thidx * 3 * poly_num * sizeof(linearpart_t), hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy working_mat_copy from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_const_mat, const_mat_arr,
			thidx * poly_num * sizeof(squarepart_t), hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy const_mat from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("enum num : %d\nblock num : %d\nthread num : %d\n", ENUM_NUM,
			BLOCK_NUM, THREAD_NUM);

	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	printf("begin solve linear system!\n");
	solveLinear<<<BLOCK_NUM, THREAD_NUM>>>(d_working_mat_copy, d_const_mat,
			d_val, d_sol_total,d_result);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch solveLinear kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	value_t h_sol_total[thidx];
	err = hipMemcpy(h_sol_total, d_sol_total, thidx * sizeof(value_t),
				hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy total solution numbers from device to host (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_result, d_result,4 * sizeof(value_t),
					hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
					"Failed to copy result from device to host (error code %s)!\n",
					hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	if(h_result[0]!=0 || h_result[1]!=0 || h_result[2]!=0 ||  h_result[3]!=0){
	    FILE *out = fopen("../data/state_files/final_messaged_state_online.txt","a+");
		printf("we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",h_result[0],h_result[1],h_result[2],h_result[3]);
			//fprintf(out,"we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",h_result[0],h_result[1],h_result[2],h_result[3]);

		tKeccakLane state[25];
		stateInit(state);
		getStates(state, var_all, h_result[0], h_result+1);

		tKeccakLane state_cp[25];
		for (int sn = 0; sn < 25; sn++) {
			state_cp[sn] = state[sn];

		}

		for (int i = 0; i < 3; i++) {
			KeccakP1600Round(state, i);
		}


		fprintf(out, "The messaged state:");
		for (int sn = 0; sn < 25; sn++) {
			if (sn % 5 == 0) {
				fprintf(out, "\n");

			}
			fprintf(out, "%016lX ", state_cp[sn]);

		}

		fprintf(out, "\n\nThe output state:");
		for (int sn = 0; sn < 25; sn++) {
			if (sn % 5 == 0) {
				fprintf(out, "\n");

			}
			fprintf(out, "%016lX ", state[sn]);

		}


		fclose(out);
		printf("finish.\n");
		exit(0);
		fclose(out);
	}else{
			FILE *out = fopen("../data/mat_files/result.txt","a+");
			long sol_all_threads = s_total_p0;
			for(int i = 0;i < thidx;i++){
				sol_all_threads += h_sol_total[i];
			}
			printf("val : %lu~%lu ,find %lu solutions, none is correct...\n",set_val << ENUM_NUM ,(set_val << ENUM_NUM)+(THREAD_NUM * BLOCK_NUM) * (1 << ENUM_NUM) -1, sol_all_threads);
			fprintf(out, "Part %d finished -- val : %lu~%lu ,find %lu solutions, none is correct...\n",atol(argv[1]), set_val << ENUM_NUM ,(set_val << ENUM_NUM)+(THREAD_NUM * BLOCK_NUM) * (1 << ENUM_NUM) -1, sol_all_threads);
			fclose(out);

	}


	printf("time:%.3lf ms\n---------------------------------------\n", msecTotal1);


	hipFree(val_arr);
	hipFree(d_working_mat_copy);
	hipFree(d_const_mat);
	hipFree(d_val);
	hipFree(d_sol_total);
	hipFree(d_result);
}
